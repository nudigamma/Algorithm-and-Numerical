#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vecAdd.cuh>
#include <cstdlib>
#include <cstdio>

/** OBJECTIVES : To run a vector Addition on the gpu .
Two input vectors are given as input and they are added in element wise fation
and they resultant per element addition is writen out on a third vector

Procedure:
Define the number of elements for the vector addition
Allocate memory for three vectors 2 in vector and one 1 out vector on CPU

Initialize the two in vectors to number and the out vectors to zero
Allocate memory for the three vectors on CPU 

Move the memory from CPU to GPU 
call the kernel and do the operation on GPU
copy back the memory from CPU to GPU 

deallocate cpu and gpu memory

**/
int main(void)
{   
    // Define default value of error 
    hipError_t err = hipSuccess;
    // Number of elements in the array
    int numElements = 50000;
    // assign the size of dynamically allocated memory
    size_t size = sizeof(float)*numElements;
    // Infor user of the vector addition operation
    std::cout << "Vector Addition of " << numElements << std::endl;

    // Allocate memory

    float * h_A = (float *) malloc(size);
    float * h_B = (float *) malloc(size);
    float * h_C = (float *) malloc(size);
    // check if pointers are pointing to memory location 
    if(h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr,"Failed to allocate host vectors !\n");
        exit(EXIT_FAILURE);
    }    
    // Initialize the host vectors

    for (int i = 0; i < numElements; i++)
    {
       h_A[i] = rand()/(float) RAND_MAX;
       h_B[i] = rand()/(float) RAND_MAX;
       // most probably assigned to 
       h_C[i] = 0;       
    }

    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Could not allocate device vector A (error code %s)",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr,"Could not allocate device vector B (error code %s)",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Couldnt not allocate devices vector C( error code %s)",hipGetErrorString (err));
        exit(EXIT_FAILURE);
    }

    printf("Copying data to device \n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Couldn't move memory from host vector A  to device vector A %s",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if(err != hipSuccess )
    {
        fprintf(stderr,"Could'nt move memory from host B  to device B %s",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    int number_of_threads = 256;

    int number_of_blocks = (int) ceil(((float)numElements)/number_of_threads);


    vectorAdd<<<number_of_blocks, number_of_threads>>> (d_A, d_B, d_C, numElements);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr,"Failure in kernel launch %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if ( err != hipSuccess)
    {
        fprintf(stderr,"Failed to copy memory from device to host %s",hipGetErrorString(err));

    }

    if(checksOut_vecAdd(h_A, h_B, h_C, numElements))
      {
          printf("vecadd on gpu validated!\n");
      }
    else 
       printf("Error exceeded tolerance!\n");

    free(h_A);
    free(h_B);
    free(h_C);

    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to free A device memory %s",hipGetErrorString(err));
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to free B device memory %s",hipGetErrorString(err));
    }

    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr,"Failed to free A device memory %s",hipGetErrorString(err));
    }

    printf("Program Terminated \n");
    return 0;
}